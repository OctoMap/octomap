
#include <hip/hip_runtime.h>
#ifdef __CUDA_SUPPORT__
#include <octomap/KeyArrayCuda.cuh>

namespace octomap {
  int KeyRayConfig::max_ray_size_ = 100;

  template <typename T>
  __device__ void ArrayCuda<T>::addKeyAtomic(const T& k) {
      ray_[atomicAdd(&last_, 1)] = k;
    }

  __device__ unsigned UnsignedArrayCuda::addKeyAtomicAt(const unsigned& k, const unsigned& l) {
    return atomicCAS((unsigned*)&ray_[l], 0, k);
  }

  template class ArrayCuda<size_t>;
  template class ArrayCuda<OcTreeKey*>;
  template class ArrayCuda<KeyHash>;
}
#endif