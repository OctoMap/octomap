
#include <hip/hip_runtime.h>
#ifdef __CUDA_SUPPORT__
#include <octomap/KeyContainerCuda.cuh>

namespace octomap {
  __device__ void KeyContainerCUDA::addKeyAtomic(const OcTreeKey& k) {
      int idx = atomicAdd(&last, 1);
      ray[idx] = k;
    }
}
#endif